#include "hip/hip_runtime.h"
#include "Test.cuh"

#define N 1

__global__ void bitreverse(unsigned int *data)
{
    unsigned int *idata = data;

    idata[threadIdx.x] = idata[threadIdx.x] * 10;
}

extern "C" float DoSomethingInCuda(float v)
{
    unsigned int *d = NULL; int i;
    unsigned int idata[N], odata[N];
    
    for (i = 0; i < N; i++)
         idata[i] = 1;

    hipMalloc((void**)&d, sizeof(int)*N);
    hipMemcpy(d, idata, sizeof(int)*N,
               hipMemcpyHostToDevice);

    bitreverse<<<1, N>>>(d);

    hipMemcpy(odata, d, sizeof(int)*N,
               hipMemcpyDeviceToHost);

    v *= odata[0];
    
    hipFree((void*)d);
    
    return v;
}