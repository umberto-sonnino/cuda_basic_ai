#include "hip/hip_runtime.h"
#include "CUDAAI.hpp"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "kernelAI.cuh"

using namespace Warped;

const double MIN_SCORE = -99999999;
template <class T>

struct KernelArray
{
    T* array;
    int size;
};

struct Result
{
    double Score;
    int Rotation;
    int Column;
};

__global__ void parallelBest(char* grid, int rows, int columns, TetrisPieces::Piece* piece, KernelArray<TetrisPieces::Piece*> upcomingPieces, int nextPieceIndex, Result *result, int mutex)
{
    //thread 1 accesses 1st element of the vector, thread 2 accesses 2nd element and so on
    
    int idx = threadIdx.x;


    if(idx < upcomingPieces.size)
    {
        double wtouch = 0;
        double wheight = 0;
        double wholes = 0;
        double wlines = 0;
        
        TetrisPieces::Piece::Definition* def = piece->GetDefinition(idx) ;
        
        for(int c = 0; c < columns; c++)
        {
            // Find collision point on this column.
            
            int collisionRow = -1;
            bool overflow = false;
            for(int r = 0; r <= rows; r++)
            {
                for(int pr = 0; pr < def->Rows; pr++)
                {
                    for(int pc = 0; pc < def->Columns; pc++)
                    {
                        if(def->Grid[pr*def->Columns+pc])
                        {
                            int gcc = c + pc;
                            int gcr = r + pr;
                            if(gcc >= columns)
                            {
                                overflow = true;
                                break;
                                continue;
                            }
                            else if(gcr >= rows || grid[gcr*columns+gcc] != -1)
                            {
                                collisionRow = r;
                                break;
                            }
                        }
                    }
                    if(collisionRow != -1)
                    {
                        break;
                    }
                }
                
                if(collisionRow != -1)
                {
                    break;
                }
            }
            
            // Found where this piece will collide on this column.
            // Determine score by lines cleared.
            collisionRow -= 1;
            
            int holesCreated = 0;
            if(collisionRow != -1 && !overflow)
            {
                double touch = 0;
                double height = 0;
                double holes = 0;
                double lines = 0;
                
                char scratchGrid[200];
                memcpy(scratchGrid, grid, rows*columns);
                char preclearGrid[200];

                
                // Commit to scratch.
                for(int pr = 0; pr < def->Rows; pr++)
                {
                    for(int pc = 0; pc < def->Columns; pc++)
                    {
                        if(def->Grid[pr*def->Columns+pc])
                        {
                            scratchGrid[(collisionRow+pr)*columns+pc+c] = 1;
                        }
                    }
                }
                                
                memcpy(preclearGrid, scratchGrid, rows*columns);

                // Check holes created
                for(int pr = 0; pr < def->Rows; pr++)
                {
                    for(int pc = 0; pc < def->Columns; pc++)
                    {
                        if(def->Grid[pr*def->Columns+pc])
                        {
                            for(int cd = collisionRow+pr+1; cd < rows; cd++)
                            {
                                if(scratchGrid[cd*columns+pc+c] == -1)
                                {
                                    holesCreated++;
                                }
                                else
                                {
                                    break;
                                }
                            }
                        }
                    }
                }
                
                // Points based on empty blocks and lines cleared.
                double positionScore = 0;//collisionRow;
                int fills = 0;
                int firstRowFill = -1;

                for(int psr = 0; psr < rows; psr++)
                {
                    for(int psc = 0; psc < columns; psc++)
                    {
                        if(scratchGrid[psr*Tetris::GridWidth+psc] != -1)
                        {
                            if(firstRowFill == -1)
                            {
                                firstRowFill = psr;
                            }
                            fills++;
                            if(psc < columns-1)
                            {
                                if(scratchGrid[psr*Tetris::GridWidth+psc+1] != -1)
                                {
                                    touch+=10;
                                }
                            }
                            if(psr == Tetris::GridHeight-1)
                            {
                                touch+=20;
                            }
                            else if(scratchGrid[(psr+1)*Tetris::GridWidth+psc] != -1)
                            {
                                touch+=10;
                            }
                        }
                    }
                }
                height = (Tetris::GridHeight-firstRowFill)*-3.5;//((double)fills/(Tetris::GridHeight-firstRowFill))*-10.0;///2.0;
                holes = holesCreated*-15.0;

                // Count lines cleared.
                int linesCleared = 0;
                for(int cr = Tetris::GridHeight-1; cr >= 0; cr--)
                {
                    int lineHasEmpty = false;
                    for(int cc = 0; cc < Tetris::GridWidth; cc++)
                    {
                        if(scratchGrid[cr*Tetris::GridWidth+cc] == -1)
                        {
                            lineHasEmpty = true;
                            break;
                        }
                    }
                    
                    // Did any lines get cleared?
                    if(!lineHasEmpty)
                    {
                        // Bump the lines above.
                        for(int kr = cr-1; kr >= 0; kr--)
                        {
                            for(int cc = 0; cc < Tetris::GridWidth; cc++)
                            {
                                scratchGrid[(kr+1)*Tetris::GridWidth+cc] = scratchGrid[kr*Tetris::GridWidth+cc];
                            }
                        }
                        
                        // Clear out top row.
                        for(int cc = 0; cc < Tetris::GridWidth; cc++)
                        {
                            scratchGrid[cc] = -1;
                        }
                        
                        // Repeat this row since we cleared it.
                        cr++;
                        
                        linesCleared++;
                    }
                }
                
                lines = linesCleared*55.0;
                positionScore = touch+height+holes+lines;
                                
                // See what the best of the rest is.
               if(false)
               {
                Result nextResult;
                nextResult.Score = MIN_SCORE;
                nextResult.Rotation = 0;
                nextResult.Column = 0;
                //1 thread for each node in the next leve of the tree.
                int numThreads = upcomingPieces.array[nextPieceIndex]->GetRotations();
                if(nextPieceIndex < upcomingPieces.size)
                {
                    int nextMutex = 0;
                    parallelBest<<<1, numThreads >>>(scratchGrid, rows, columns, upcomingPieces.array[nextPieceIndex], upcomingPieces, nextPieceIndex+1, &nextResult, nextMutex);
                }
                __syncthreads();
                while(mutex != 0);//wait
                mutex = 1;
                if(positionScore + nextResult.Score > result->Score)
                {
                    result->Score = positionScore + nextResult.Score;
                    result->Rotation = idx;
                    result->Column = c;
                }
               }
               else{
                if(positionScore > result->Score)
                {
                    
                    wtouch = touch;
                    wheight = height;
                    wholes = holes;
                    wlines = lines;
                    
                    result->Score = positionScore;
                    result->Rotation = idx;
                    result->Column = c;
                }
                mutex = 0;
               }
            }
            
        }
    }
}


extern void GetCUDAMoves(char* grid, const int gridRows, const int gridColumns, int currentRow, int currentColumn, int currentRotation, 
    TetrisPieces::Piece* currentPiece, std::vector<TetrisPieces::Piece*> upcomingPieces, std::vector<TetrisMove> *moves)
{
    static char scratchGrid[Tetris::GridWidth*Tetris::GridHeight];
    memcpy(scratchGrid, grid, gridRows*gridColumns);
    
    Result result;
    result.Score = MIN_SCORE;
    result.Rotation = 0;
    result.Column = 0;
    KernelArray<TetrisPieces::Piece*> kernel_Array;
    
    kernel_Array.array = (TetrisPieces::Piece**) malloc(sizeof(TetrisPieces::Piece)*upcomingPieces.size());
    kernel_Array.size = upcomingPieces.size();
    
    for(int i = 0; i < upcomingPieces.size(); i++)
    {
        TetrisPieces::Piece* piece = upcomingPieces[i];
        kernel_Array.array[i] = piece;
        
    }
    //1 thread for every rotation & ~1 block for every piece, since 4 max rotations on any piece
    int numThreads = currentPiece->GetRotations();
    
    
    char *deviceGrid;
    hipMalloc(&deviceGrid, gridRows*gridColumns);   
    hipMemcpy(deviceGrid, (void*)grid, gridRows*gridColumns, hipMemcpyHostToDevice);

    Result *deviceResult;
    hipMalloc(&deviceResult, sizeof(result));
    hipMemcpy(deviceResult, (void*)&result, sizeof(result), hipMemcpyHostToDevice);

    TetrisPieces::Piece *devicePiece;
    hipMalloc(&devicePiece, sizeof(currentPiece));
    hipMemcpy(devicePiece, (void*)currentPiece, sizeof(currentPiece), hipMemcpyHostToDevice);

    parallelBest <<<1, numThreads>>> (deviceGrid, gridRows, gridColumns, devicePiece, kernel_Array, 0, deviceResult, 0);

    
    result.Rotation = deviceResult->Rotation;
    result.Column = deviceResult->Column;
    result.Score = deviceResult->Score;

    // Do rotation moves.
    int r = currentRotation;
    while(r != result.Rotation)
    {
        moves->push_back(TetrisMove(TetrisMoveType(6)/*::RotateRight*/));
        r = (r+1)%currentPiece->GetRotations();
    }
  
    // Do column moves.
    while(currentColumn < result.Column)
    {
        moves->push_back(TetrisMove(TetrisMoveType(1)/*::MoveRight*/));
        currentColumn++;
    }
    while(currentColumn > result.Column)
    {
        moves->push_back(TetrisMove(TetrisMoveType(0)/*::MoveLeft*/));
        currentColumn--;
    }
    
    moves->push_back(TetrisMove(TetrisMoveType(4)/*::SlamDown*/));
}